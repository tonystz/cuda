extern "C" {

 
#include <hip/hip_runtime.h>
#include <stdlib.h>

 const int COL_NUM=2383;
 const int ROW_NUM=479;
 const int LOG_LEN=44;
 const int THRESHOLD=500;
 
 /* for test
 const int COL_NUM=40;
 const int ROW_NUM=2;
 const int LOG_LEN=20;*/
    typedef struct{
      char ipAddr[LOG_LEN];
      int cnt;
    }ST_IPAddr;
    __device__ void mystrcpy(char *out,char *in){
      int i=0;
      for(i=0;in[i]!=0;i++){
        out[i] = in[i];
      }
      out[i]=0;
    }
__device__ char* myitoa(int i, char b[]){
    char const digit[] = "0123456789";
    char* p = b;
    if(i<0){
        *p++ = '-';
        i *= -1;
    }
    int shifter = i;
    do{ //Move to where representation ends
        ++p;
        shifter = shifter/10;
    }while(shifter);
    *p = '\0';
    do{ //Move back, inserting digits as u go
        *--p = digit[i%10];
        i = i/10;
    }while(i);
    return b;
}

__device__ int myAtoi(char* str)
{
    // Initialize result
    int res = 0;
 
    for (int i = 0; str[i] != '\0'; ++i)
        res = res * 10 + str[i] - '0';
 
    // return result.
    return res;
}

 __device__ int splitStrInt(char *s, char addr[]){
    int i=0;
    while(s[i] !=0){
        if(s[i]=='#')break;
        addr[i]=s[i];
        i++;
    }
    addr[i]=0;
    return myAtoi(s+i+1);
 }
 __device__ char* mergStrInt(char*s, int n, char a[]){
    int i=0;
    while(s[i]!=0){
        a[i]=s[i];
        i++;
    }
    a[i]='#';
    i++;
    char buff[512];
    myitoa(n,buff);
    int j=0;
    while(buff[j] !=0){
        a[i+j]=buff[j];
        j++;
    }
    a[i+j]=0;
    return a;
 }
    __device__  int find_404_ipaddr(char*in_gpu,char outAddr[],int rowStartIndex ,int strStartIndex){
        int index=0;
        for(int i = 0; i<LOG_LEN; i++){
            index = strStartIndex+i+rowStartIndex;
            if (in_gpu[index]=='-'){
              if (in_gpu[index+1] == '4' && in_gpu[index+2] == '0' && in_gpu[index+3] == '4'){
                 outAddr[i]=0;
                 return 1;
              }
            }
            outAddr[i]=in_gpu[index];
        }
        return 0;
    }
   
    __device__ int strEqua(char *s1,char *s2){
      if((*s1==0)&&(*s2 !=0)|| (*s1 !=0 && *s2==0))return 0;
      for(int i=0;s1[i]!=0 && s2[i] !=0;i++){
        if (s1[i] != s2[i]){
          return 0;
        }
      }
      return 1;
    }
    __device__ void pushDedup(ST_IPAddr*dedup,char *ipAddr,int*dedup_cnt){
      int c=0;
      for(;c<COL_NUM;c++){
        if(strEqua(dedup[c].ipAddr,ipAddr)){
          dedup[c].cnt ++;
          break;
        }else{
          if(dedup[c].ipAddr[0] == 0){
            mystrcpy(dedup[c].ipAddr,ipAddr);
            dedup[c].cnt = 1;
            (*dedup_cnt)++;
            break;
          }
        }
      }
    }

    __device__  ST_IPAddr *getHeapMem(){
      /*
      https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html?highlight=dynamic#heap-memory-allocation
      need set the limit, so hear for ROW_NUM=40
      */ 
      int mem_size=ROW_NUM*COL_NUM*sizeof(ST_IPAddr);
      ST_IPAddr *heap = (ST_IPAddr *)malloc(mem_size);

      memset(heap, 0, mem_size);
      printf("Thread %d got pointer: %p heapsize=%d\n", threadIdx.x, heap,mem_size);
      //free(heap);
      return heap;
    }

    __device__ void showHeap(ST_IPAddr *heap,int dedup_cnt){
      //print heap memory
      printf("Thread:%d Total:%d\n", threadIdx.x, dedup_cnt);
      for(int i=0;i<dedup_cnt;i++){
        if(heap[i].cnt>THRESHOLD){
            printf("[showHeap]Thread %d addr: %s=%d\n", threadIdx.x, heap[i].ipAddr,heap[i].cnt);
        }
        
      }
    }

    __device__ void pushDedupSummary(ST_IPAddr*dedup, ST_IPAddr *st,int*dedup_cnt){
      int c=0;
      for(;c<COL_NUM;c++){
        if(strEqua(dedup[c].ipAddr,st->ipAddr)){
          dedup[c].cnt += st->cnt;
          break;
        }else{
          if(dedup[c].ipAddr[0] == 0){
            mystrcpy(dedup[c].ipAddr,st->ipAddr);
            dedup[c].cnt = st->cnt;
            (*dedup_cnt)++;
            break;
          }
        }
      }
    }

    __device__ void summary(char*out_gpu, ST_IPAddr *heap){
      //int idx = threadIdx.x + blockIdx.x * blockDim.x;
      int rowOffset=COL_NUM*LOG_LEN;
      char tmpBuff[LOG_LEN];
      int heap_dedup_cnt=0;
       for(int r=0;r<ROW_NUM;r++){
         for(int c=0;c<COL_NUM;c++){
           if(out_gpu[r*rowOffset+c*LOG_LEN]!=0){
              //printf("thread[%d]offset=%d r=%d,c=%d  find:%s\n",idx,r*rowOffset+c*LOG_LEN,r,c,out_gpu+r*rowOffset+c*LOG_LEN);
              
              //de-dedup global
              
              ST_IPAddr st;
              st.cnt=splitStrInt(out_gpu+r*rowOffset+c*LOG_LEN,tmpBuff);
              mystrcpy(st.ipAddr,tmpBuff);
              pushDedupSummary(heap,&st,&heap_dedup_cnt);
           }
         }
       }

       showHeap(heap,heap_dedup_cnt);

    }

    __global__ void check_log(char*in_gpu,char* out_gpu)
    { 
      ST_IPAddr dedup[COL_NUM];
      int dedup_cnt=0;    
      
      int idx = threadIdx.x + blockIdx.x * blockDim.x;
      // printf("ST_IPAddr size: %lu\n",sizeof(ST_IPAddr));
      //printf("index: threadIdx.x=%d blockIdx.x=%d  blockDim.x=%d \n",threadIdx.x,blockIdx.x,blockDim.x);
      //printf("thread id: [%d]  get str:%s  strlen=%d \n",idx,in_gpu+LOG_LEN+1, LOG_LEN);

      // string len is 6
      char sub[LOG_LEN+1];
      int strStartIndex=0;//string start locaion
      int rowOffset=COL_NUM*LOG_LEN;
      if(idx > ROW_NUM){
        printf("[ERROR]idx=%d, rowCnt=%d, we want one thread process all one rows'data .please tune",idx,ROW_NUM);
      }
      
      int rowStartIndex=rowOffset*idx;
      out_gpu[rowStartIndex]=0;
      //int all_find_iadd_in_row=0;
      for(int c=0;c<COL_NUM;c++){
        strStartIndex=c*LOG_LEN;
        if (find_404_ipaddr(in_gpu,sub,rowStartIndex,strStartIndex)){
          //printf("thread[%d] rowStartIndex=%d, substring:%s strStartIndex %d\n",idx,rowStartIndex,sub,strStartIndex);
          
          //copy 404 to out_gpu
          //mystrcpy(out_gpu+rowStartIndex+dedup_cnt*LOG_LEN,sub);
          //mystrcpy(dedup[c].ipAddr,sub);
          pushDedup(dedup,sub,&dedup_cnt);
          //printf("rowStartIndex=%d,c=%d  sub=%s add=%d dedup_cnt=%d \n",rowStartIndex,c,sub,dedup[c].cnt,dedup_cnt);
          
        }
      }

      //int rowOffsetForInt=COL_NUM*4;
      __syncthreads();
      char tmpBuff[LOG_LEN*2];
      for(int c=0;c<dedup_cnt;c++){
         mergStrInt(dedup[c].ipAddr,dedup[c].cnt,tmpBuff);
         mystrcpy(out_gpu+rowStartIndex+LOG_LEN*c,tmpBuff);
         //memcpy(out_int_gpu+idx*rowOffsetForInt+c*4, &dedup[c], sizeof(ST_IPAddr));
         //out_int_gpu[idx*rowOffsetForInt+c*4]=dedup[c].cnt;
         //printf("thread[%d] rowStartIndex[%d][%d] addr:%s = %d\n",idx,rowStartIndex,c,dedup[c].ipAddr,dedup[c].cnt);
      }
      //printf("thread[%d] addrCnt:%d\n",idx,dedup_cnt);
      __syncthreads();

      if(idx ==0 ){
        // in global mem to dedup
        ST_IPAddr *heap=getHeapMem();
        summary(out_gpu,heap);
        if(heap)free(heap);
      }
    }
    
}
